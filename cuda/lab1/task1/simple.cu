#include "hip/hip_runtime.h"
// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple

#include <stdio.h>

const int N = 16; 
const int blocksize = 16; 

__global__ 
void simple(float *c) 
{
	c[threadIdx.x] = threadIdx.x;
}

int main()
{
	float *c = new float[N];	
	float *cd;
	const int size = N*sizeof(float);
	
	hipMalloc( (void**)&cd, size );
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	simple<<<dimGrid, dimBlock>>>(cd);
	hipDeviceSynchronize();
	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 
	hipFree( cd );
	
	for (int i = 0; i < N; i++)
		printf("%f ", c[i]);
	printf("\n");
	delete[] c;
	printf("done\n");
	return EXIT_SUCCESS;
}
