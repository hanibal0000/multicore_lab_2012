#include "hip/hip_runtime.h"
// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple

#include <stdio.h>
#include <math.c>

const int N = 16; 
const int blocksize = 16; 

__global__ 
void simple(float *c) 
{
	c[threadIdx.x] = sqrt(c[threadIdx.x]);
}

int main()
{
	float *res = new float[N];	
    float *orig = new float[N];
    float *test = new float[N];
	float *cd;
	const int size = N*sizeof(float);
	
	hipMalloc( (void**)&cd, size );
    int i;
    elem = 4;
    for(i = 0; i<N; i++)
    {
        orig[i] = elem;
        test[i] = sqrt(elem);
        elem = pow(elem, 2);
    }

    hipMemcpy( cd, orig, size, hipMemcpyHostToDevice );
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	simple<<<dimGrid, dimBlock>>>(cd);
	hipDeviceSynchronize();
	hipMemcpy( res, cd, size, hipMemcpyDeviceToHost ); 
	hipFree( cd );
	
	for (i = 0; i < N; i++)
		printf("Input: %f, CPU: %f, CUDA: %f ", orig[i], test[i], res[i]);
	printf("\n");
	delete[] c;
	printf("done\n");
	return EXIT_SUCCESS;
}
