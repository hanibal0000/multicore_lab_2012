#include "hip/hip_runtime.h"
// Ingemars rewrite of the julia demo, integrating the OpenGL parts.
// The CUDA parts are - intentionally - NOT rewritten, and have some
// serious performance problems. Find the problems and make this a�
// decently performing CUDA program.

// Compile with
// nvcc -lglut -lGL interactiveJulia.cu -o interactiveJulia

#include <GL/glut.h>
#include <GL/gl.h>
#include <stdio.h>

// Image data
	unsigned char	*pixels;
	int	 gImageWidth, gImageHeight;

// Init image data
void initBitmap(int width, int height)
{
	pixels = (unsigned char *)malloc(width * height * 4);
	gImageWidth = width;
	gImageHeight = height;
}

#define DIM 1024

// Complex number class
struct hipComplex
{
    float   r;
    float   i;
    
    __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
    
    __device__ float magnitude2( void )
    {
        return r * r + i * i;
    }
    
    __device__ hipComplex operator*(const hipComplex& a)
    {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    
    __device__ hipComplex operator+(const hipComplex& a)
    {
        return hipComplex(r+a.r, i+a.i);
    }
};

__device__ int julia( int x, int y, float r, float im)
{
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

//    hipComplex c(-0.8, 0.156);
    hipComplex c(r, im);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++)
    {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return i;
    }

    return i;
}

__global__ void kernel( unsigned char *ptr, float r, float im)
{
    // map from blockIdx to pixel position
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    // now calculate the value at that position
    int juliaValue = julia( x, y, r, im );
    ptr[offset*4 + 0] = 255 * juliaValue/200;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;
}

float theReal, theImag;

// Compute CUDA kernel and display image
void Draw()
{
	unsigned char *dev_bitmap;
	
	hipMalloc( &dev_bitmap, gImageWidth*gImageHeight*4 );

	dim3	grid(DIM,DIM);
	kernel<<<grid,1>>>( dev_bitmap, theReal, theImag);
	hipDeviceSynchronize();
	hipMemcpy( pixels, dev_bitmap, gImageWidth*gImageHeight*4, hipMemcpyDeviceToHost );
	
	hipFree( dev_bitmap );
	
// Dump the whole picture onto the screen.	
	glClearColor( 0.0, 0.0, 0.0, 1.0 );
	glClear( GL_COLOR_BUFFER_BIT );
	glDrawPixels( gImageWidth, gImageHeight, GL_RGBA, GL_UNSIGNED_BYTE, pixels );
	glutSwapBuffers();
}

void MouseMovedProc(int x, int y)
{
	theReal = -0.5 + (float)(x-400) / 500.0;
	theImag = -0.5 + (float)(y-400) / 500.0;
	printf("real = %f, imag = %f\n", theReal, theImag);
	glutPostRedisplay ();
}

// Main program, inits
int main( int argc, char** argv) 
{
	glutInit(&argc, argv);
	glutInitDisplayMode( GLUT_DOUBLE | GLUT_RGBA );
	glutInitWindowSize( DIM, DIM );
	glutCreateWindow("CUDA on live GL");
	glutDisplayFunc(Draw);
	glutPassiveMotionFunc(MouseMovedProc);
	
	initBitmap(DIM, DIM);
	
	glutMainLoop();
}
