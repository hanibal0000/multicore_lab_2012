// Matrix addition, CPU version
// gcc matrix_cpu.c -o matrix_cpu -std=c99


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void add_matrix(float *in, float *in_out)
{
    int index = threadIdx.y * blockDim.x + threadIdx.x;
    if(index < blockDim.x * blockDim.y)
        in_out[index] = in[index] + in_out[index];
    
}

int main()
{
	const int N = 16;

	float *a = new float[N*N];
	float *b = new float[N*N];
	float *cu_in, *cu_out;
    size_t size = N*N*sizeof(float);

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}

    hipMalloc((void **)&cu_in, size); 
    hipMalloc((void **)&cu_out, size); 
    hipMemcpy(cu_in, a, size, hipMemcpyHostToDevice);
    hipMemcpy(cu_out, b, size, hipMemcpyHostToDevice);

    dim3 dimBlock(N,N);
    dim3 dimGrid(1,1);

    add_matrix<<<dimGrid, dimBlock>>>(cu_in, cu_out);

    hipMemcpy(b, cu_out, size, hipMemcpyDeviceToHost);
	
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%0.2f ", b[i+j*N]);
		}
		printf("\n");
	}
}
