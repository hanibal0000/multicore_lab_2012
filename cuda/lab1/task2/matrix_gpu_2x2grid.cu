#include "hip/hip_runtime.h"
// Matrix addition, CPU version
// gcc matrix_cpu.c -o matrix_cpu -std=c99

#include <stdio.h>

#define MATRIX_SIZE 16
#define GRID_DIM 2

__global__
void add_matrix(float *in, float *in_out)
{
    //number of blocks * length of one block line
    int line_length = gridDim.x * blockDim.x;
    //height of a block * position of block + position of thread in block
    int line_number = blockDim.y  * blockIdx.y + threadIdx.y;
    //length of a block * position of block + position of thread
    int line_pos = blockDim.x * blockIdx.x + threadIdx;

    int index = (line_number - 1) * line_length + line_pos;

    if(index < (gridDim.x * blockDim.x) * (gridDim.y * blockDim.y))
        in_out[index] = in[index] + in_out[index];
    
}

int main()
{
    
	const int N = MATRIX_SIZE;

	float *a = new float[N*N];
	float *b = new float[N*N];
	float *cu_in, *cu_out;
    size_t size = N*N*sizeof(float);

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}

    hipMalloc((void **)&cu_in, size); 
    hipMalloc((void **)&cu_out, size); 
    hipMemcpy(cu_in, a, size, hipMemcpyHostToDevice);
    hipMemcpy(cu_out, b, size, hipMemcpyHostToDevice);

    dim3 dimBlock(N/GRID_DIM,N/GRID_DIM);
    dim3 dimGrid(GRID_DIM,GRID_DIM);

    add_matrix<<<dimGrid, dimBlock>>>(cu_in, cu_out);

    hipMemcpy(b, cu_out, size, hipMemcpyDeviceToHost);
	
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%0.2f ", b[i+j*N]);
		}
		printf("\n");
	}
}
